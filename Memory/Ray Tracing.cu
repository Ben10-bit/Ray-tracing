#include "hip/hip_runtime.h"
//ʹ�ó����ڴ����ĳЩȫ���ڴ棬�������
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "../common/cpu_bitmap.h"

#define rnc(x) (x * rand() / RAND_MAX)
#define SPHERES 20
#define INF 2e9
#define DIM 1024

struct Sphere
{
	float r, g, b;
	float x, y, z;
	float radius;
	
	__device__  float hit(int ox, int oy, float* n)
	{
		float dx = ox - x;
		float dy = oy - y;

		float t = sqrtf(dx * dx + dy * dy);
		if (t < radius)
		{
			float dz = sqrtf(radius * radius - t * t);
			*n = dz / sqrtf(radius * radius);
			return dz + z;
		}

		return -INF;
	}
};

//���ó����ڴ�
__constant__ Sphere s[SPHERES];


__global__ void kernel(unsigned char* ptr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int ox = x - DIM / 2;
	int oy = y - DIM / 2;

	float r = 0, g = 0, b = 0;
	float maxz = -INF;
	for (int i = 0; i < SPHERES; i++)
	{
		float n;
		float t = s[i].hit(ox, oy, &n);
		if (t > maxz)
		{
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;

			maxz = t;
		}
	}

	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}



int main()
{
	//�����¼����ڼ�ʱ
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	CPUBitmap bitmap(DIM, DIM);
	unsigned char* dev_bitmap;

	hipMalloc((void**)&dev_bitmap, bitmap.image_size());

	//������ʱ��������ʼ��������GPUֵ
	Sphere* temp = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
	for (int i = 0; i < SPHERES; i++)
	{
		temp[i].r = rnc(1.0f);
		temp[i].g = rnc(1.0f);
		temp[i].b = rnc(1.0f);
		temp[i].x = rnc(1000.0f) - 500;
		temp[i].y = rnc(1000.0f) - 500;
		temp[i].z = rnc(1000.0f) - 500;
		temp[i].radius = rnc(100.0f) + 20;
  	}
	hipMemcpyToSymbol(HIP_SYMBOL(s), temp, sizeof(Sphere) * SPHERES);
	free(temp);

	//�涨�̸߳��С
	dim3 grid(DIM / 16, DIM / 16);
	dim3 block(16, 16);

	kernel << <grid, block >> > (dev_bitmap);

	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

	//ֹͣ��ʱ
	hipEventRecord(stop, 0);
	//ʱ��ͬ�����ȵ�stopǰ����䶼ִ���꣬Ȼ���ִ�н����������
	hipEventSynchronize(stop);

	//����ʱ��
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Time to generate = %3.1f ms\n", elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	//չʾͼ��
	bitmap.display_and_exit();

	hipFree(dev_bitmap);

	return 0;
}
